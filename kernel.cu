#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__ void kernelVerificador(int n, int* array_rand, int* array_result) {
    int i = threadIdx.x;
    while (i < n) {
        array_result[i] = array_rand[i] % 2;
        i += blockDim.x;
    }
}

int main(){
    int* arrRand, * arrResult;
    int* d_arrRand, * d_arrResult; 
    
    int n = 10;
    
    int tamanho = n * sizeof(int); 

    arrRand = (int*)malloc(tamanho);
    arrResult = (int*)malloc(tamanho);

    hipMalloc((void**)&d_arrRand, tamanho);
    hipMalloc((void**)&d_arrResult, tamanho);

    int maior = 9;
    int menor = 1;

    for (int i = 0; i < n; i++) {
        arrRand[i] = rand() % maior + menor;
    }

    hipMemcpy(d_arrRand, arrRand, tamanho, hipMemcpyHostToDevice);

    kernelVerificador << <1, 10 >> > (n, d_arrRand, d_arrResult);

    hipMemcpy(arrResult, d_arrResult, tamanho, hipMemcpyDeviceToHost);

    printf("Resultado da verificacao: \n");
    for (int i = 0; i < n; i++) {
        printf("%d | %d \n", arrRand[i], arrResult[i]);
    }

    hipFree(d_arrRand); 
    hipFree(d_arrResult);
    
    return 0;
}